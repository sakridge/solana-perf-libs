#include "hip/hip_runtime.h"
#include "fe.cu"
#include "ge.cu"
#include "gpu_common.h"
#define USE_CLOCK_GETTIME
#include "perftime.h"

bool g_verbose = true;

#define SCALAR_SIZE 32
__global__ void ge3_kernel(
        const unsigned char* a, //32-byte
        uint8_t* out,
        uint64_t total
        ) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < total) {
        ge_p3 r;
        ge_scalarmult_base(&r, &a[i * SCALAR_SIZE]);
        out[i] = r.X[0] ^ r.Y[1] ^ r.Z[7];
    }
}

int main(int argc, const char* argv[]) {
    if (argc < 3) {
        printf("invalid args\n");
        return 1;
    }
    int arg = 1;
    uint64_t total = strtol(argv[arg++], NULL, 10);
    int num_threads_per_block  = strtol(argv[arg++], NULL, 10);

    size_t ab_size = SCALAR_SIZE * total;
    unsigned char* a_h = (unsigned char*)calloc(ab_size, 1);
    unsigned char* out_h = (unsigned char*)calloc(total, 1);
    size_t p3_size = sizeof(ge_p3) * total;

    for (uint i = 0; i < ab_size; i++) {
        a_h[i] = rand();
    }

    unsigned char* a_d = NULL;
    uint8_t* out_d = NULL;

    CUDA_CHK(hipMalloc(&a_d, ab_size));
    CUDA_CHK(hipMalloc(&out_d, total));

    hipStream_t stream = {};
    CUDA_CHK(hipStreamCreate(&stream));

    CUDA_CHK(hipMemcpyAsync(a_d, a_h, ab_size, hipMemcpyHostToDevice, stream));

    int num_blocks = ROUND_UP_DIV(total, num_threads_per_block);

    perftime_t start, end;
    get_time(&start);
    //for (int i = 0; i < 10; i++) {
        ge3_kernel<<<num_blocks, num_threads_per_block, 0, stream>>>
                (a_d,
                 out_d,
                 total);
        CUDA_CHK(hipPeekAtLastError());
    //}

    hipError_t err = hipMemcpyAsync(out_h, out_d, total, hipMemcpyDeviceToHost, stream);
    CUDA_CHK(err);

    CUDA_CHK(hipStreamSynchronize(stream));

    get_time(&end);
    LOG("time diff: %.2f us\n", get_diff(&start, &end));

    CUDA_CHK(hipFree(a_d));
    CUDA_CHK(hipFree(out_d));
    free(a_h);
    free(out_h);
}
